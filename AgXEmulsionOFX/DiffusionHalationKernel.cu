#include "hip/hip_runtime.h"
// GPU diffusion & halation (very naive reference implementation)
#include <hip/hip_runtime.h>
#include "DiffusionHalationKernel.cuh"

#define MAX_RADIUS 10

__device__ __forceinline__ int clampi(int v, int lo, int hi) {return v < lo ? lo : (v > hi ? hi : v);} 

__global__ void DiffusionHalationKernel(float* img, int width, int height, int stride, int rad, float halStrength)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= width || y >= height) return;

    int index = (y * width + x) * 4; // RGBA float

    // compute blurred red component (box blur)
    float sumR = 0.f;
    int count = 0;
    for(int dy=-rad; dy<=rad; ++dy){
        int yy = clampi(y + dy, 0, height-1);
        for(int dx=-rad; dx<=rad; ++dx){
            int xx = clampi(x + dx, 0, width-1);
            int idx = (yy * width + xx) * 4;
            sumR += img[idx]; // red
            count++;
        }
    }
    float blurR = sumR / (float)count;

    // original red
    float origR = img[index];
    float newR = origR + halStrength * (blurR - origR);
    img[index] = fminf(fmaxf(newR, 0.f), 1.f);
    // diffusion: simple overall blur of all channels (optional small)
    // Using same box blur scaled by radius*0.1 for now - placeholder
    // Could do proper Gaussian later
}

extern "C" void LaunchDiffusionHalationCUDA(float* img, int width, int height, float radius, float halStrength)
{
    if(radius < 0.5f && halStrength < 1e-5f) return; // skip if negligible
    int rad = (int)(radius + 0.5f);
    if(rad < 1) rad = 1;
    if(rad > MAX_RADIUS) rad = MAX_RADIUS;

    dim3 block(16,16);
    dim3 grid((width + block.x -1)/block.x, (height + block.y -1)/block.y);
    DiffusionHalationKernel<<<grid, block>>>(img, width, height, width*4, rad, halStrength);
    hipDeviceSynchronize();
} 