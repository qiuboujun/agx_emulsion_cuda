#include "hip/hip_runtime.h"
// GPU diffusion & halation (matching Python implementation exactly)
#include <hip/hip_runtime.h>
#include "DiffusionHalationKernel.cuh"

#define MAX_RADIUS 25

__device__ __forceinline__ int clampi(int v, int lo, int hi) {return v < lo ? lo : (v > hi ? hi : v);} 

__global__ void DiffusionHalationKernel(float* img, int width, int height, int stride, int rad, float halStrength)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= width || y >= height) return;

    int index = (y * width + x) * 4; // RGBA float

    // Gaussian blur on all channels (diffusion effect)
    float sigma = rad * 0.5f + 1e-3f;
    float twoSigma2 = 2.0f * sigma * sigma;
    float sumR = 0.f, sumG = 0.f, sumB = 0.f;
    float wsum = 0.f;
    
    for(int dy=-rad; dy<=rad; ++dy){
        int yy = clampi(y + dy, 0, height-1);
        for(int dx=-rad; dx<=rad; ++dx){
            int xx = clampi(x + dx, 0, width-1);
            float w = __expf(-(dx*dx + dy*dy)/twoSigma2);
            int idx = (yy * width + xx) * 4;
            sumR += w * img[idx];     // R channel
            sumG += w * img[idx + 1]; // G channel  
            sumB += w * img[idx + 2]; // B channel
            wsum += w;
        }
    }
    
    // Normalize blurred values
    float blurR = sumR / wsum;
    float blurG = sumG / wsum;
    float blurB = sumB / wsum;

    // Apply diffusion to all channels
    img[index] = blurR;     // R channel (diffusion)
    img[index + 1] = blurG; // G channel (diffusion)
    img[index + 2] = blurB; // B channel (diffusion)

    // Apply halation effect to red channel only (matching Python implementation)
    float origR = img[index];
    float newR = origR + halStrength * (blurR - origR);
    img[index] = fminf(fmaxf(newR, 0.f), 1.f); // Clamp to [0,1]
}

extern "C" void LaunchDiffusionHalationCUDA(float* img, int width, int height, float radius, float halStrength)
{
    if(radius < 0.5f && halStrength < 1e-5f) return; // skip if negligible
    int rad = (int)(radius + 0.5f);
    if(rad < 1) rad = 1;
    if(rad > MAX_RADIUS) rad = MAX_RADIUS;

    dim3 block(16,16);
    dim3 grid((width + block.x -1)/block.x, (height + block.y -1)/block.y);
    DiffusionHalationKernel<<<grid, block>>>(img, width, height, width*4, rad, halStrength);
    hipDeviceSynchronize();
} 