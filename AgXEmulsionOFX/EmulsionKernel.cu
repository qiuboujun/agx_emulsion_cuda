#include "hip/hip_runtime.h"
#include "EmulsionKernel.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>

__constant__ float c_logE[601];
__constant__ float c_curveR[601];
__constant__ float c_curveG[601];
__constant__ float c_curveB[601];
__constant__ float c_gamma;
__constant__ float c_exposureEV;

__device__ __forceinline__ float lerp(float a,float b,float t){return a+(b-a)*t;}

__device__ float lookupDensity(const float* curve,const float* logE,float val){
    // clamp to table range
    if(val <= logE[0]) {
        return curve[0];
    }
    if(val >= logE[600]) {
        return curve[600];
    }
    // linear search (601 small)
    int idx=0;
    for(int i=1;i<601;i++){if(val<logE[i]){idx=i-1;break;}}
    float t=(val-logE[idx])/(logE[idx+1]-logE[idx]);
    return lerp(curve[idx],curve[idx+1],t);
}

__global__ void EmulsionKernel(float* img, int width, int height)
{
    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    int total=width*height;
    if(idx>=total) return;
    float4* pix = reinterpret_cast<float4*>(img);
    float4 p = pix[idx];
    
    // Debug center pixel only
    bool isCenter = (idx == (height/2) * width + width/2);
    
    // Input p.x, p.y, p.z are CMY light values from Camera LUT stage
    // We need to convert them to log exposure values first
    // CMY light values are in the range [0, ~10] typically
    // Log exposure should be in the range [-3, 4] typically
    
    // Convert CMY light values to log exposure
    // The relationship is: log_exposure = log10(CMY_light)
    float logER = log10f(fmaxf(p.x,1e-6f));
    float logEG = log10f(fmaxf(p.y,1e-6f));
    float logEB = log10f(fmaxf(p.z,1e-6f));
    
    // Apply exposure adjustment
    logER += c_exposureEV*0.30103f;
    logEG += c_exposureEV*0.30103f;
    logEB += c_exposureEV*0.30103f;
    
    if(isCenter) {
        printf("Emulsion DEBUG: Input CMY=(%f,%f,%f)\n", p.x, p.y, p.z);
        printf("Emulsion DEBUG: Log exposure=(%f,%f,%f)\n", logER, logEG, logEB);
        printf("Emulsion DEBUG: After gamma=(%f,%f,%f)\n", logER/c_gamma, logEG/c_gamma, logEB/c_gamma);
        printf("Emulsion DEBUG: Film curve range=[%f,%f]\n", c_logE[0], c_logE[600]);
    }
    
    // Convert log-exposure to CMY density using film curves
    float densityR = lookupDensity(c_curveR, c_logE, logER / c_gamma);
    float densityG = lookupDensity(c_curveG, c_logE, logEG / c_gamma);
    float densityB = lookupDensity(c_curveB, c_logE, logEB / c_gamma);
    
    if(isCenter) {
        printf("Emulsion DEBUG: Lookup results=(%f,%f,%f)\n", densityR, densityG, densityB);
    }
    
    p.x = densityR;
    p.y = densityG;
    p.z = densityB;
    
    pix[idx]=p;
}

extern "C" void LaunchEmulsionCUDA(float* img, int width, int height, float gamma, float exposureEV)
{
    int total = width * height;
    int block = 256;
    int grid = (total + block - 1) / block;
    hipMemcpyToSymbol(HIP_SYMBOL(c_gamma),&gamma,sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_exposureEV),&exposureEV,sizeof(float));
    EmulsionKernel<<<grid, block>>>(img, width, height);
    hipDeviceSynchronize();
}

extern "C" void UploadLUTCUDA(const float* logE,const float* r,const float* g,const float* b){
    hipMemcpyToSymbol(HIP_SYMBOL(c_logE),logE,601*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_curveR),r,601*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_curveG),g,601*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_curveB),b,601*sizeof(float));
} 