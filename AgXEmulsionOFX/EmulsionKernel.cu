#include "hip/hip_runtime.h"
#include "EmulsionKernel.cuh"
#include <hip/hip_runtime.h>

__constant__ float c_logE[601];
__constant__ float c_curveR[601];
__constant__ float c_curveG[601];
__constant__ float c_curveB[601];
__constant__ float c_gamma;
__constant__ float c_exposureEV;

__device__ __forceinline__ float lerp(float a,float b,float t){return a+(b-a)*t;}

__device__ float lookupDensity(const float* curve,const float* logE,float val){
    // clamp to table range
    if(val <= logE[0]) {
        return curve[0];
    }
    if(val >= logE[600]) {
        return curve[600];
    }
    // linear search (601 small)
    int idx=0;
    for(int i=1;i<601;i++){if(val<logE[i]){idx=i-1;break;}}
    float t=(val-logE[idx])/(logE[idx+1]-logE[idx]);
    return lerp(curve[idx],curve[idx+1],t);
}

__global__ void EmulsionKernel(float* img, int width, int height)
{
    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    int total=width*height;
    if(idx>=total) return;
    float4* pix = reinterpret_cast<float4*>(img);
    float4 p = pix[idx];
    float logER = log10f(fmaxf(p.x,1e-6f)) + c_exposureEV*0.30103f;
    float logEG = log10f(fmaxf(p.y,1e-6f)) + c_exposureEV*0.30103f;
    float logEB = log10f(fmaxf(p.z,1e-6f)) + c_exposureEV*0.30103f;
    // Convert log-exposure to CMY density
    p.x = lookupDensity(c_curveR, c_logE, logER / c_gamma);
    p.y = lookupDensity(c_curveG, c_logE, logEG / c_gamma);
    p.z = lookupDensity(c_curveB, c_logE, logEB / c_gamma);
    pix[idx]=p;
}

extern "C" void LaunchEmulsionCUDA(float* img, int width, int height, float gamma, float exposureEV)
{
    int total = width * height;
    int block = 256;
    int grid = (total + block - 1) / block;
    hipMemcpyToSymbol(HIP_SYMBOL(c_gamma),&gamma,sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_exposureEV),&exposureEV,sizeof(float));
    EmulsionKernel<<<grid, block>>>(img, width, height);
    hipDeviceSynchronize();
}

extern "C" void UploadLUTCUDA(const float* logE,const float* r,const float* g,const float* b){
    hipMemcpyToSymbol(HIP_SYMBOL(c_logE),logE,601*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_curveR),r,601*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_curveG),g,601*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_curveB),b,601*sizeof(float));
} 